#include <iostream>
#include <cmath>
#include <chrono>

#include "hip/hip_runtime.h"

// function to retrieve cuda errors
// inspired from:
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuCheck(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Kernel function to add the elements of two arrays
__global__
void add(std::size_t numElements, float* a, float* b, float* result) {
  for (std::size_t i = 0; i < numElements; i++) {
    result[i] = a[i] + b[i]; 
  }
} 

int main(void)
{
  int N = 1<<20;

  // Allocate memory on the host
  float* vec1 = new float[N];
  float* vec2 = new float[N];
  float* vec3 = new float[N];

  // initialize x and y arrays on the host
  for (std::size_t i = 0; i < N; i++) {
    vec1[i] = 1.0f;
    vec2[i] = 2.0f;
    vec3[i] = 0.0f;
  }

  // Allocate device memory
  float* vec1_d;
  float* vec2_d;
  float* vec3_d;

  auto start = std::chrono::system_clock::now();

  gpuCheck(hipMallocManaged(&vec1_d, N * sizeof(float)), __FILE__, __LINE__);
  gpuCheck(hipMallocManaged(&vec2_d, N * sizeof(float)), __FILE__, __LINE__);
  gpuCheck(hipMallocManaged(&vec3_d, N * sizeof(float)), __FILE__, __LINE__);

  // Transfer data from host to device memory
  gpuCheck(hipMemcpy(vec1_d, vec1, N * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__);
  gpuCheck(hipMemcpy(vec2_d, vec2, N * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__);
  gpuCheck(hipMemcpy(vec3_d, vec3, N * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__);

  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, vec1_d, vec2_d, vec3_d);

  // Wait for GPU to finish before accessing on host
  gpuCheck(hipDeviceSynchronize(), __FILE__, __LINE__);

  // Copy the result to host memory
  gpuCheck(hipMemcpy(vec3, vec3_d, N * sizeof(float), hipMemcpyDeviceToHost), __FILE__, __LINE__);

  auto end = std::chrono::system_clock::now();
  auto elapsed = end - start;

  std::cout << "cuda single thread add duration: " << elapsed.count() << " nanoseconds" << std::endl;


  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = fmax(maxError, fabs(vec3[i] - 3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;

  // Free host memory
  delete [] vec1;
  delete [] vec2;
  delete [] vec3;

  // Free device memory
  hipFree(vec1_d);
  hipFree(vec2_d);
  hipFree(vec3_d);
  
  return 0;
}